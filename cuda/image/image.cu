#include "hip/hip_runtime.h"
#include "image.hpp"
#include "cuda_utils.hpp"

#include <string> // std::string

#include "stb_image.h" // stbi_load

void ImageProgram::setInitialState()
{
  std::string filename = "textures/lena512.png";
  int channels;
  auto h_pixels = stbi_load(filename.c_str(), &extent.x, &extent.y, &channels,
    STBI_rgb_alpha);
  if (!h_pixels)
  {
    printf("failed to load texture image");
    return;
  }
  auto tex_size = sizeof(uchar4) * extent.x * extent.y;
  checkCuda(hipMemcpy(d_pixels, h_pixels, tex_size, hipMemcpyHostToDevice));
  stbi_image_free(h_pixels);
}

void ImageProgram::cleanup()
{
  checkCuda(hipStreamSynchronize(stream));
  checkCuda(hipStreamDestroy(stream));
  checkCuda(hipFree(d_pixels));
}

ImageProgram::ImageProgram()
{
  checkCuda(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
}

void ImageProgram::runTimestep() {}
