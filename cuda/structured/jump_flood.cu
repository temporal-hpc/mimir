#include "hip/hip_runtime.h"
#include "jump_flood.hpp"
#include "cuda_utils.hpp"

#include <limits> // std::numeric_limits

constexpr float max_distance = std::numeric_limits<float>::max();

__device__
float4 jumpFloodStep(float2 coord, float4 *seeds, int step_length, int2 extent)
{
  float best_dist = max_distance;
  float2 best_coord = make_float2(-1.f, -1.f);

  for (int y = -1; y <= 1; ++y)
  {
    for (int x = -1; x <= 1; ++x)
    {
      int sample_x = coord.x + x * step_length;
      int sample_y = coord.y + y * step_length;
      if (sample_x >= 0 && sample_x < extent.x && sample_y >= 0 && sample_y < extent.y)
      {
        float4 seed = seeds[extent.x * sample_y + sample_x];
        float dist = hypotf(seed.x - coord.x, seed.y - coord.y);

        if ((seed.x != -1.f && seed.y != -1.f) && dist < best_dist)
        {
          best_dist = dist;
          best_coord = make_float2(seed.x, seed.y);
        }
      }
    }
  }
  return make_float4(best_coord.x, best_coord.y, 0.f, best_dist);
}

__global__
void kernelJfa(float4 *result, float4 *seeds, const int2 extent, int step_length)
{
  const int tx = blockDim.x * blockIdx.x + threadIdx.x;
  const int ty = blockDim.y * blockIdx.y + threadIdx.y;
  if (tx < extent.x && ty < extent.y)
  {
    float2 coord = make_float2(tx, ty);
    float4 output = jumpFloodStep(coord, seeds, step_length, extent);
    result[extent.x * ty + tx] = output;
  }
}

__global__
void kernelDistanceTransform(float *distances, float4 *seeds, int2 extent)
{
  const int tx = blockDim.x * blockIdx.x + threadIdx.x;
  const int ty = blockDim.y * blockIdx.y + threadIdx.y;

  if (tx < extent.x && ty < extent.y)
  {
    auto grid_idx = extent.x * ty + tx;
    distances[grid_idx] = seeds[grid_idx].w / hypotf(extent.x, extent.y);
  }
}

void jumpFlood(float *distances, float4 *seeds[], int2 extent, hipStream_t stream)
{
  dim3 threads(32, 32);
  dim3 blocks( (extent.x + threads.x - 1) / threads.x,
               (extent.y + threads.y - 1) / threads.y );

  int out_idx = 0, in_idx = 1;
  for (int k = extent.x / 2; k > 0; k = k >> 1)
  {
    kernelJfa<<< blocks, threads, 0, stream >>>(
      seeds[out_idx], seeds[in_idx], extent, k
    );
    checkCuda(hipDeviceSynchronize());
    std::swap(out_idx, in_idx);
  }
  kernelDistanceTransform<<< blocks, threads, 0, stream >>>(
    distances, seeds[in_idx], extent
  );
  checkCuda(hipDeviceSynchronize());
}

__global__
void kernelSetNonSeeds(float4 *seeds, int seed_count)
{
  int tx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tx < seed_count)
  {
    seeds[tx] = make_float4(-1.f, -1.f, 0.f, 0.f);
  }
}

__global__
void kernelSetSeeds(float4 *seeds, float *raw_coords,
  int coord_count, int2 extent)
{
  int tx = blockIdx.x * blockDim.x + threadIdx.x;

  if (tx < coord_count)
  {
    auto coord = reinterpret_cast<float2*>(raw_coords)[tx];
    int2 point{ (int)coord.x, (int)coord.y };
    if (point.x >= 0 && point.x < extent.x && point.y >= 0 && point.y < extent.y)
    {
      seeds[extent.x * point.y + point.x] = make_float4(coord.x, coord.y, 0.f, 0.f);
    }
  }
}

void initJumpFlood(float4 *d_seeds, float *d_coords, int coord_count,
  int2 extent, hipStream_t stream)
{
  dim3 threads{128};
  dim3 blocks1{ (extent.x * extent.y + threads.x - 1) / threads.x};
  dim3 blocks2{ (coord_count + threads.x - 1) / threads.x};

  kernelSetNonSeeds<<< blocks1, threads, 0, stream >>>(d_seeds, extent.x * extent.y);
  checkCuda(hipStreamSynchronize(stream));
  kernelSetSeeds<<< blocks2, threads, 0, stream >>>(
    d_seeds, d_coords, coord_count, extent
  );
  checkCuda(hipStreamSynchronize(stream));
}

__global__ void initSystem(float *coords, size_t particle_count,
  hiprandState *global_states, int2 extent, unsigned seed)
{
  auto particles = reinterpret_cast<float2*>(coords);
  auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
  if (tidx < particle_count)
  {
    auto local_state = global_states[tidx];
    hiprand_init(seed, tidx, 0, &local_state);
    auto rx = extent.x * hiprand_uniform(&local_state);
    auto ry = extent.y * hiprand_uniform(&local_state);
    float2 p{rx, ry};
    particles[tidx] = p;
    global_states[tidx] = local_state;
  }
}

__device__ float clamp(float x, float low, float high)
{
  return fmaxf(low, fminf(high, x));
}

__global__ void integrate2d(float *coords, size_t particle_count,
  hiprandState *global_states, int2 extent)
{
  auto particles = reinterpret_cast<float2*>(coords);
  auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
  if (tidx < particle_count)
  {
    auto local_state = global_states[tidx];
    auto r = hiprand_normal2(&local_state);
    auto p = particles[tidx];
    p.x = clamp(p.x + r.x, 1e-6f, extent.x);
    p.y = clamp(p.y + r.y, 1e-6f, extent.y);
    particles[tidx] = p;
    global_states[tidx] = local_state;
  }
}

JumpFloodProgram::JumpFloodProgram(size_t point_count, int width, int height):
  element_count{point_count}, extent{width, height}
{
  checkCuda(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
}

void JumpFloodProgram::setInitialState()
{
  checkCuda(hipSetDevice(0));

  //checkCuda(hipMalloc(&_d_distances, dist_size));
  //checkCuda(hipMalloc(&d_coords, sizeof(float2) * element_count));
  checkCuda(hipMallocAsync(&d_states, sizeof(hiprandState) * element_count, stream));

  dim3 threads{128};
  dim3 blocks { (element_count + threads.x - 1) / threads.x};
  initSystem<<<blocks, threads>>>(d_coords, element_count, d_states, extent, 1234);
  checkCuda(hipDeviceSynchronize());

	// Allocate device numeric canvas
	size_t seed_sizes = sizeof(float4) * extent.x * extent.y;
	checkCuda(hipMalloc(&d_grid[0], seed_sizes));
  checkCuda(hipMalloc(&d_grid[1], seed_sizes));
  checkCuda(hipDeviceSynchronize());
	initJumpFlood(d_grid[1], d_coords, element_count, extent, stream);
}

void JumpFloodProgram::cleanup()
{
  checkCuda(hipDeviceSynchronize());
  checkCuda(hipStreamDestroy(stream));
  checkCuda(hipFree(d_grid[0]));
  checkCuda(hipFree(d_grid[1]));
  checkCuda(hipFree(d_states));
  checkCuda(hipFree(d_distances));
  checkCuda(hipFree(d_coords));
  checkCuda(hipDeviceReset());
}

void JumpFloodProgram::runTimestep()
{
  dim3 threads{128};
  dim3 blocks { (element_count + threads.x - 1) / threads.x};

  integrate2d<<< blocks, threads, 0, stream >>>(
    d_coords, element_count, d_states, extent
  );
  checkCuda(hipDeviceSynchronize());

  initJumpFlood(d_grid[1], d_coords, element_count, extent, stream);

  jumpFlood(d_distances, d_grid, extent, stream);
}
