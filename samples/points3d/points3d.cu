#include "hip/hip_runtime.h"
#include <cudaview/vk_engine.hpp>

#include <hiprand/hiprand_kernel.h>

#include <string> // std::stoul
#include <cudaview/validation.hpp>
using namespace validation; // checkCuda

__global__ void initSystem(float *coords, size_t point_count,
    hiprandState *global_states, uint3 extent, unsigned seed)
{
    auto points = reinterpret_cast<float3*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < point_count)
    {
        auto local_state = global_states[tidx];
        hiprand_init(seed, tidx, 0, &local_state);
        auto rx = extent.x * hiprand_uniform(&local_state);
        auto ry = extent.y * hiprand_uniform(&local_state);
        auto rz = extent.z * hiprand_uniform(&local_state);
        points[tidx] = {rx, ry, rz};
        global_states[tidx] = local_state;
    }
}

__global__ void integrate3d(float *coords, size_t point_count,
    hiprandState *global_states, uint3 extent)
{
    auto points = reinterpret_cast<float3*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < point_count)
    {
        auto local_state = global_states[tidx];
        auto p = points[tidx];
        p.x += hiprand_normal(&local_state);
        if (p.x > extent.x) p.x = extent.x;
        if (p.x < 0) p.x = 0;
        p.y += hiprand_normal(&local_state);
        if (p.y > extent.x) p.y = extent.y;
        if (p.y < 0) p.y = 0;
        p.z += hiprand_normal(&local_state);
        if (p.z > extent.z) p.z = extent.z;
        if (p.z < 0) p.z = 0;
        points[tidx] = p;
        global_states[tidx] = local_state;
    }
}

int main(int argc, char *argv[])
{
    float *d_coords       = nullptr;
    hiprandState *d_states = nullptr;
    unsigned block_size   = 256;
    unsigned seed         = 123456;
    uint3 extent          = {200, 200, 200};

    // Default values for this program
    size_t point_count = 100;
    size_t iter_count  = 10000;
    int width = 1280;
    int height = 720;
    if (argc >= 3) { width = std::stoi(argv[1]); height = std::stoi(argv[2]); }
    if (argc >= 4) point_count = std::stoul(argv[3]);
    if (argc >= 5) iter_count = std::stoul(argv[4]);

    bool display = true;
    if (width == 0 || height == 0)
    {
        width = height = 10;
        display = false;
    }

    ViewerOptions options;
    options.window = {width,height}; // Starting window size
    options.show_metrics = false; // Show metrics window in GUI
    options.report_period = 30; // Print relevant usage stats every N seconds
    VulkanEngine engine;
    engine.init(options);

    if (display)
    {
        ViewParams params;
        params.element_count = point_count;
        params.element_size = sizeof(float3);
        params.extent = extent;
        params.data_domain = DataDomain::Domain3D;
        params.resource_type = ResourceType::UnstructuredBuffer;
        params.primitive_type = PrimitiveType::Points;
        engine.createView((void**)&d_coords, params);
    }
    else // Run the simulation without display
    {
        checkCuda(hipMalloc((void**)&d_coords, sizeof(float3) * point_count));
    }

    checkCuda(hipMalloc(&d_states, sizeof(hiprandState) * point_count));
    unsigned grid_size = (point_count + block_size - 1) / block_size;
    initSystem<<<grid_size, block_size>>>(d_coords, point_count, d_states, extent, seed);
    checkCuda(hipDeviceSynchronize());

    if (display) engine.displayAsync();
    for (size_t i = 0; i < iter_count; ++i)
    {
        if (i == iter_count / 2) engine.showMetrics();
        if (display) engine.prepareWindow();
        integrate3d<<<grid_size, block_size>>>(d_coords, point_count, d_states, extent);
        checkCuda(hipDeviceSynchronize());
        if (display) engine.updateWindow();
    }

    checkCuda(hipFree(d_states));
    checkCuda(hipFree(d_coords));

    return EXIT_SUCCESS;
}
