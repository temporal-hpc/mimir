#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream> // std::cerr
#include <string> // std::stoul

#include <mimir/mimir.hpp>
#include <mimir/validation.hpp> // checkCuda
using namespace mimir;
using namespace mimir::validation; // checkCuda

__global__
void initSystem(double2 *coords, double *sizes, size_t point_count,
    hiprandState *global_states, int2 extent, unsigned seed)
{
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < point_count)
    {
        auto local_state = global_states[tidx];
        hiprand_init(seed, tidx, 0, &local_state);
        auto rx = extent.x * hiprand_uniform_double(&local_state);
        auto ry = extent.y * hiprand_uniform_double(&local_state);
        // Generate a point size up to 10;
        double2 p{rx, ry};
        coords[tidx] = p;
        sizes[tidx]  = 10 * hiprand_uniform_double(&local_state);
        global_states[tidx] = local_state;
    }
}

__global__
void integrate2d(double2 *coords, size_t point_count, hiprandState *states, int2 extent)
{
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < point_count)
    {
        auto local_state = states[tidx];
        auto r = hiprand_normal2_double(&local_state);
        auto p = coords[tidx];
        p.x += r.x;
        if (p.x > extent.x) p.x = extent.x;
        if (p.x < 0) p.x = 0;
        p.y += r.y;
        if (p.y > extent.y) p.y = extent.y;
        if (p.y < 0) p.y = 0;
        coords[tidx] = p;
        states[tidx] = local_state;
    }
}

int main(int argc, char *argv[])
{
    size_t point_count    = 100;
    size_t iter_count     = 10000;
    double2 *d_coords     = nullptr;
    double *d_sizes        = nullptr;
    hiprandState *d_states = nullptr;
    int2 extent           = {200, 200};
    unsigned block_size   = 256;
    unsigned grid_size    = (point_count + block_size - 1) / block_size;
    unsigned seed         = 123456;

    if (argc >= 2) point_count = std::stoul(argv[1]);
    if (argc >= 3) iter_count = std::stoul(argv[2]);
    try
    {
        // Initialize engine
        ViewerOptions options;
        options.window_size = {1920,1080}; // Starting window size
        options.present = PresentOptions::VSync;
        MimirEngine engine;
        engine.init(options);

        MemoryParams m;
        m.layout          = DataLayout::Layout1D;
        m.element_count.x = point_count;
        m.component_type  = ComponentType::Double;
        m.channel_count   = 2;
        m.resource_type   = ResourceType::Buffer;
        auto points = engine.createBuffer((void**)&d_coords, m);

        m.component_type  = ComponentType::Double;
        m.channel_count   = 1;
        auto sizes = engine.createBuffer((void**)&d_sizes, m);

        ViewParams params;
        params.element_count = point_count;
        params.extent        = {200, 200, 1};
        params.data_domain   = DataDomain::Domain2D;
        params.domain_type   = DomainType::Unstructured;
        params.view_type     = ViewType::Markers;
        params.options.default_size = 20.f;
        /*params.options.external_shaders = {
            {"shaders/marker_vertexMain.spv", VK_SHADER_STAGE_VERTEX_BIT},
            {"shaders/marker_geometryMain.spv", VK_SHADER_STAGE_GEOMETRY_BIT},
            {"shaders/marker_fragmentMain.spv", VK_SHADER_STAGE_FRAGMENT_BIT}
        };*/
        params.attributes[AttributeType::Position] = *points;
        params.attributes[AttributeType::Size] = *sizes;
        engine.createView(params);

        // Cannot make CUDA calls that use the target device memory before
        // registering it on the engine
        //checkCuda(hipMalloc(&d_coords, sizeof(double2) * point_count));
        checkCuda(hipMalloc(&d_states, sizeof(hiprandState) * point_count));
        initSystem<<<grid_size, block_size>>>(
            d_coords, d_sizes, point_count, d_states, extent, seed
        );
        checkCuda(hipDeviceSynchronize());

        // Set up the cuda code that updates the view buffer as a lambda function
        auto cuda_call = [&]
        {
            integrate2d<<< grid_size, block_size >>>(
                d_coords, point_count, d_states, extent
            );
            checkCuda(hipDeviceSynchronize());
        };
        // Start rendering loop with the above function
        engine.display(cuda_call, iter_count);
    }
    catch (const std::exception& e)
    {
        std::cerr << e.what() << std::endl;
    }

    checkCuda(hipFree(d_states));
    checkCuda(hipFree(d_coords));
    checkCuda(hipFree(d_sizes));

    return EXIT_SUCCESS;
}
