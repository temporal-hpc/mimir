#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream> // std::cerr
#include <string> // std::stoul

#include <mimir/mimir.hpp>
#include "validation.hpp" // checkCuda
using namespace mimir;

__global__
void initSystem(double2 *coords, double *sizes, size_t point_count,
    hiprandState *global_states, int2 extent, unsigned seed)
{
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < point_count)
    {
        auto local_state = global_states[tidx];
        hiprand_init(seed, tidx, 0, &local_state);
        auto rx = extent.x * hiprand_uniform_double(&local_state);
        auto ry = extent.y * hiprand_uniform_double(&local_state);
        // Generate a point size up to 10;
        double2 p{rx, ry};
        coords[tidx] = p;
        sizes[tidx]  = 10 * hiprand_uniform_double(&local_state);
        global_states[tidx] = local_state;
    }
}

__global__
void integrate2d(double2 *coords, size_t point_count, hiprandState *states, int2 extent)
{
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < point_count)
    {
        auto local_state = states[tidx];
        auto r = hiprand_normal2_double(&local_state);
        auto p = coords[tidx];
        p.x += r.x;
        if (p.x > extent.x) p.x = extent.x;
        if (p.x < 0) p.x = 0;
        p.y += r.y;
        if (p.y > extent.y) p.y = extent.y;
        if (p.y < 0) p.y = 0;
        coords[tidx] = p;
        states[tidx] = local_state;
    }
}

int main(int argc, char *argv[])
{
    unsigned point_count  = 100;
    unsigned iter_count   = 10000;
    double2 *d_coords     = nullptr;
    double *d_sizes       = nullptr;
    hiprandState *d_states = nullptr;
    int2 extent           = {200, 200};
    unsigned block_size   = 256;
    unsigned grid_size    = (point_count + block_size - 1) / block_size;
    unsigned seed         = 123456;

    if (argc >= 2) point_count = std::stoul(argv[1]);
    if (argc >= 3) iter_count  = std::stoul(argv[2]);

    // Initialize engine
    ViewerOptions options;
    options.window.size  = {1920,1080}; // Starting window size
    options.present.mode = PresentMode::VSync;

    EngineHandle engine = nullptr;
    createEngine(options, &engine);

    AllocHandle points = nullptr, sizes = nullptr;
    allocLinear(engine, (void**)&d_coords, sizeof(double2) * point_count, &points);
    allocLinear(engine, (void**)&d_sizes, sizeof(double) * point_count, &sizes);

    ViewHandle view = nullptr;
    ViewDescription desc{
        .element_count = point_count,
        .view_type     = ViewType::Markers,
        .domain_type   = DomainType::Domain2D,
        .extent        = ViewExtent::make(200, 200, 1),
        .attributes    = {
            { AttributeType::Position, {
                .source     = points,
                .size       = point_count,
                .format     = FormatDescription::make<double2>(),
                .indices    = nullptr,
                .index_size = 0,
            }},
            { AttributeType::Size, {
                .source     = sizes,
                .size       = point_count,
                .format     = FormatDescription::make<double>(),
                .indices    = nullptr,
                .index_size = 0,
            }},
        }
    };
    createView(engine, &desc, &view);

    // Cannot make CUDA calls that use the target device memory before
    // registering it on the engine
    //checkCuda(hipMalloc(&d_coords, sizeof(double2) * point_count));
    checkCuda(hipMalloc(&d_states, sizeof(hiprandState) * point_count));
    initSystem<<<grid_size, block_size>>>(
        d_coords, d_sizes, point_count, d_states, extent, seed
    );
    checkCuda(hipDeviceSynchronize());

    // Set up the cuda code that updates the view buffer as a lambda function
    auto cuda_call = [&]
    {
        integrate2d<<< grid_size, block_size >>>(d_coords, point_count, d_states, extent);
        checkCuda(hipDeviceSynchronize());
    };
    // Start rendering loop with the above function
    display(engine, cuda_call, iter_count);

    checkCuda(hipFree(d_states));
    checkCuda(hipFree(d_coords));
    checkCuda(hipFree(d_sizes));
    destroyEngine(engine);

    return EXIT_SUCCESS;
}
