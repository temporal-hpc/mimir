#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream> // std::cerr
#include <string> // std::stoul

#include <mimir/cudaview.hpp>
#include <mimir/validation.hpp> // checkCuda
using namespace mimir;
using namespace mimir::validation; // checkCuda

struct CudaProgram
{
    hipStream_t stream   = nullptr;
    float *d_coords       = nullptr;
    size_t particle_count = 0;
    hiprandState *d_states = nullptr;
    int2 bounding_box     = {0, 0};
    unsigned block_size   = 256;
    unsigned grid_size    = 0;
    size_t state_count    = 0;
    unsigned seed         = 0;

    CudaProgram(size_t particle_count, int width, int height, unsigned seed = 0);
    void setInitialState();
    void cleanup();
    void runTimestep();
};

__global__ void initSystem(float *coords, size_t particle_count,
    hiprandState *global_states, size_t state_count, int2 extent, unsigned seed)
{   
    auto particles = reinterpret_cast<float2*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < particle_count)
    {
        auto local_state = global_states[tidx];
        hiprand_init(seed, tidx, 0, &local_state);
        auto rx = extent.x * hiprand_uniform(&local_state);
        auto ry = extent.y * hiprand_uniform(&local_state);
        float2 p{rx, ry};
        particles[tidx] = p;
        global_states[tidx] = local_state;
    }
}

__global__ void integrate2d(float *coords, size_t particle_count,
    hiprandState *global_states, size_t state_count, int2 extent)
{
    auto particles = reinterpret_cast<float2*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < particle_count)
    {
        auto local_state = global_states[tidx];
        auto r = hiprand_normal2(&local_state);
        auto p = particles[tidx];
        p.x += r.x;
        if (p.x > extent.x) p.x = extent.x;
        if (p.x < 0) p.x = 0;
        p.y += r.y;
        if (p.y > extent.y) p.y = extent.y;
        if (p.y < 0) p.y = 0;
        particles[tidx] = p;
        global_states[tidx] = local_state;
    }
}

CudaProgram::CudaProgram(size_t particle_count, int width, int height, unsigned seed):
    particle_count(particle_count), bounding_box{width, height},
    state_count(particle_count), seed(seed),
    grid_size((particle_count + block_size - 1) / block_size)
{
    checkCuda(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
}

void CudaProgram::setInitialState()
{
    //checkCuda(hipMalloc(&d_coords, sizeof(float2) * particle_count));
    checkCuda(hipMalloc(&d_states, sizeof(hiprandState) * state_count));
    initSystem<<<grid_size, block_size>>>(
        d_coords, particle_count, d_states, state_count, bounding_box, seed
    );
    //checkCuda(hipDeviceSynchronize());
}

void CudaProgram::cleanup()
{
    checkCuda(hipStreamDestroy(stream));
    checkCuda(hipFree(d_states));
    checkCuda(hipFree(d_coords));
}

void CudaProgram::runTimestep()
{
    integrate2d<<< grid_size, block_size, 0, stream >>>(
        d_coords, particle_count, d_states, state_count, bounding_box
    );
    //checkCuda(hipStreamSynchronize(stream));
}

int main(int argc, char *argv[])
{
    size_t particle_count = 100;
    size_t iter_count = 10000;
    if (argc >= 2)
    {
        particle_count = std::stoul(argv[1]);
    }
    if (argc >= 3)
    {
        iter_count = std::stoul(argv[2]);
    }

    CudaProgram program(particle_count, 200, 200, 123456);
    try
    {
        // Initialize engine
        ViewerOptions options;
        options.window_size = {1920,1080}; // Starting window size
        options.present = PresentOptions::VSync;
        CudaviewEngine engine;
        engine.init(options);
        ViewParams params;
        params.element_count = program.particle_count;
        params.extent = {200, 200, 1};
        params.data_type = DataType::Float;
        params.channel_count = 2;
        params.data_domain = DataDomain::Domain2D;
        params.resource_type = ResourceType::UnstructuredBuffer;
        params.element_type = ElementType::Markers;
        params.hip_stream = program.stream;
        params.options.size = 20.f;
        /*params.options.external_shaders = {
            {"shaders/marker_vertexMain.spv", VK_SHADER_STAGE_VERTEX_BIT},
            {"shaders/marker_geometryMain.spv", VK_SHADER_STAGE_GEOMETRY_BIT},
            {"shaders/marker_fragmentMain.spv", VK_SHADER_STAGE_FRAGMENT_BIT}
        };*/
        engine.createView((void**)&program.d_coords, params);

        // Cannot make CUDA calls that use the target device memory before
        // registering it on the engine
        program.setInitialState();

        // Set up the function that we want to display
        auto timestep_function = std::bind(&CudaProgram::runTimestep, program);
        // Start rendering loop
        engine.display(timestep_function, iter_count);
    }
    catch (const std::exception& e)
    {
        std::cerr << e.what() << std::endl;
        program.cleanup();
        return EXIT_FAILURE;
    }
    program.cleanup();

    return EXIT_SUCCESS;
}
