#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <omp.h>
#define CA_LOW 2
#define CA_HIGH 3
#define CA_NACER 3

#include <iostream>

#include "tools.h"
#include "kernel3D.cuh"
#include "openmp3D.h"

#include <mimir/mimir.hpp>
#include <mimir/validation.hpp> // checkCuda
using namespace mimir;
using namespace mimir::validation; // checkCuda

int main(int argc, char **argv){
    if(argc != 8){
        fprintf(stderr, "ejecutar como ./prog n nt B seed steps prob modo\nmodo = 0 CPU,  1 GPU\nB <= 10 (blocksize is BxBxB)\n\n");
        exit(EXIT_FAILURE);
    }
    const char *map[2] = {"CPU", "GPU"};
    // args
    long n       = atoi(argv[1]);
    int nt      = atoi(argv[2]);
    int B       = atoi(argv[3]);
    int seed    = atoi(argv[4]);
    int steps   = atoi(argv[5]);
    float prob  = atof(argv[6]);
    int modo = atoi(argv[7]);
    float timems;
    double t1;

    if(B > 10 || modo > 1){
        fprintf(stderr, "ejecutar como ./prog n nt B seed steps prob modo\nmodo = 0 CPU,  1 GPU\nB <= 10 (blocksize is BxBxB)\n\n");
        exit(EXIT_FAILURE);
    }

    // SETEO DE OPENMP THREADS (solo relevante para inicializar datos y solucion CPU)

    omp_set_num_threads(nt);
    // TODO CAMBIAR A 2D
    printf("modo: %s     n=%ld (%.3f GiBytes / cubo)    nt=%i   B=%i  steps=%i\n", map[modo], n, sizeof(int)*n*n*n/(1024*1024*1024.0), nt, B, steps);

    // original (3D)
    // TODO CAMBIAR A 2D
    int *original = new int[n*n*n];

    // punteros GPU (3D)
    int *d1, *d2;

    // CREACION DE DATOS
    printf("Inicializando.................."); fflush(stdout);
    t1 = omp_get_wtime();
    init_prob(n, original, seed, prob);

    int width = 1920, height = 1080;
    MimirEngine engine;
    engine.init(width, height);

    MemoryParams mp;
    mp.layout         = DataLayout::Layout3D;
    mp.element_count  = {(uint)n, (uint)n, (uint)n};
    mp.component_type = ComponentType::Int;
    mp.channel_count  = 1;
    mp.resource_type  = ResourceType::Buffer;
    //mp.resource_type  = ResourceType::LinearTexture;
    auto m1 = engine.createBuffer((void**)&d1, mp);
    auto m2 = engine.createBuffer((void**)&d2, mp);

    ViewParams params;
    params.element_count = n * n * n;
    params.extent        = {(unsigned)n, (unsigned)n, (unsigned)n};
    params.data_domain   = DataDomain::Domain3D;
    params.domain_type   = DomainType::Structured;
    params.view_type     = ViewType::Voxels;
    //params.view_type     = ViewType::Image;
    params.attributes[AttributeType::Color] = *m1;
    params.options.default_size = 5.f;
    /*params.options.external_shaders = {
        {"shaders/voxel_vertexImplicitMain.spv", VK_SHADER_STAGE_VERTEX_BIT},
        {"shaders/voxel_geometryMain.spv", VK_SHADER_STAGE_GEOMETRY_BIT},
        {"shaders/voxel_fragmentMain.spv", VK_SHADER_STAGE_FRAGMENT_BIT}
    };*/
    /*params.options.external_shaders = {
        {"shaders/texture_vertex3dMain.spv", VK_SHADER_STAGE_VERTEX_BIT},
        {"shaders/texture_frag3d_Float1.spv", VK_SHADER_STAGE_FRAGMENT_BIT}
    };*/
    auto v1 = engine.createView(params);

    params.attributes[AttributeType::Color] = *m2;
    params.options.visible = false;
    auto v2 = engine.createView(params);

    // TODO CAMBIAR A 2D
    gpuErrchk(hipMemcpy(d1, original, sizeof(int)*n*n*n, hipMemcpyHostToDevice));
    printf("done: %f secs\n", omp_get_wtime() - t1);

    engine.displayAsync();

    // ejecucion
    print_cube(n, original, "INPUT");
    printf("Press Enter...\n"); fflush(stdout); getchar();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // OJO: la cantidad total de threads tiene que ser Bx * By * Bz <= 1024
    // TODO CAMBIAR A 2D
    dim3 block(B,B,B);
    // TODO CAMBIAR A 2D
    dim3 grid((n+block.x-1)/block.x, (n+block.y-1)/block.y, (n+block.z-1)/block.z);
    if(modo==1){
        // modo GPU
        for(int i=0; i<steps; ++i){
            printf("[GPU] Simulacion step=%i........", i);
            hipEventRecord(start);

            // llamada al kernel
            engine.prepareViews();

            kernel_CA3D<<<grid, block>>>(n, d1, d2);
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );

            v1->toggleVisibility();
            v2->toggleVisibility();

            engine.updateViews();

            // tiempo y print
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&timems, start, stop);
            printf("done: %f\n", timems/1000.0);
            // TODO CAMBIAR A 2D
            gpuErrchk(hipMemcpy(original, d2, sizeof(int)*n*n*n, hipMemcpyDeviceToHost));
            print_cube(n, original, "[GPU] Automata celular");
            printf("Press Enter...\n"); fflush(stdout); getchar();
            std::swap(d1, d2);
        }
    }
    else{
        // secundario CPU (3D)
        // TODO CAMBIAR A 2D
        int *CPUd2 = new int[n*n*n];

        // modo CPU (multicore segun nt escogido)
        for(int i=0; i<steps; ++i){
            printf("[CPU] Simulacion step=%i........", i);
            t1 = omp_get_wtime();

            // llamada a paso de simulacion
            openmp_CA3D(n, original, CPUd2);

            // tiempo y print
            printf("done: %f\n", omp_get_wtime() - t1);
            print_cube(n, CPUd2, "Automata celular (CPU):");
            printf("Press Enter...\n"); fflush(stdout); getchar();
            std::swap(original, CPUd2);
        }
    }
    printf("Finished running all steps\n");
}
