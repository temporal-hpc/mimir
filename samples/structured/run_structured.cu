#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <limits> // std::numeric_limits

#include <mimir/mimir.hpp>
#include <mimir/validation.hpp> // checkCuda
using namespace mimir;
using namespace mimir::validation; // checkCuda

constexpr float max_distance = std::numeric_limits<float>::max();

struct JumpFloodProgram
{
    hipStream_t stream    = nullptr;
    float *d_distances     = nullptr;
    float *d_coords        = nullptr;
    float4 *d_grid[2]      = {nullptr, nullptr};
    hiprandState *d_states  = nullptr;
    unsigned element_count = 0;
    int2 extent            = {0, 0};

    JumpFloodProgram(unsigned particle_count, int width, int height);
    void setInitialState();
    void cleanup();
    void runTimestep();
};

__device__
float4 jumpFloodStep(float2 coord, float4 *seeds, int step_length, int2 extent)
{
    float best_dist = max_distance;
    float2 best_coord = make_float2(-1.f, -1.f);

    for (int y = -1; y <= 1; ++y)
    {
        for (int x = -1; x <= 1; ++x)
        {
        int sample_x = coord.x + x * step_length;
        int sample_y = coord.y + y * step_length;
        if (sample_x >= 0 && sample_x < extent.x && sample_y >= 0 && sample_y < extent.y)
        {
            float4 seed = seeds[extent.x * sample_y + sample_x];
            float dist = hypotf(seed.x - coord.x, seed.y - coord.y);

            if ((seed.x != -1.f && seed.y != -1.f) && dist < best_dist)
            {
            best_dist = dist;
            best_coord = make_float2(seed.x, seed.y);
            }
        }
        }
    }
    return make_float4(best_coord.x, best_coord.y, 0.f, best_dist);
}

__global__
void kernelJfa(float4 *result, float4 *seeds, const int2 extent, int step_length)
{
    const int tx = blockDim.x * blockIdx.x + threadIdx.x;
    const int ty = blockDim.y * blockIdx.y + threadIdx.y;
    if (tx < extent.x && ty < extent.y)
    {
        float2 coord = make_float2(tx, ty);
        float4 output = jumpFloodStep(coord, seeds, step_length, extent);
        result[extent.x * ty + tx] = output;
    }
}

__global__
void kernelDistanceTransform(float *distances, float4 *seeds, int2 extent)
{
    const int tx = blockDim.x * blockIdx.x + threadIdx.x;
    const int ty = blockDim.y * blockIdx.y + threadIdx.y;

    if (tx < extent.x && ty < extent.y)
    {
        auto grid_idx = extent.x * ty + tx;
        distances[grid_idx] = seeds[grid_idx].w / hypotf(extent.x, extent.y);
    }
}

void jumpFlood(float *distances, float4 *seeds[], int2 extent, hipStream_t stream)
{
    dim3 threads(32, 32);
    dim3 blocks( (extent.x + threads.x - 1) / threads.x,
                (extent.y + threads.y - 1) / threads.y );

    int out_idx = 0, in_idx = 1;
    for (int k = extent.x / 2; k > 0; k = k >> 1)
    {
        kernelJfa<<< blocks, threads, 0, stream >>>(
        seeds[out_idx], seeds[in_idx], extent, k
        );
        checkCuda(hipDeviceSynchronize());
        std::swap(out_idx, in_idx);
    }
    kernelDistanceTransform<<< blocks, threads, 0, stream >>>(
        distances, seeds[in_idx], extent
    );
    checkCuda(hipDeviceSynchronize());
}

__global__
void kernelSetNonSeeds(float4 *seeds, int seed_count)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if (tx < seed_count)
    {
        seeds[tx] = make_float4(-1.f, -1.f, 0.f, 0.f);
    }
}

__global__
void kernelSetSeeds(float4 *seeds, float *raw_coords,
    int coord_count, int2 extent)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if (tx < coord_count)
    {
        auto coord = reinterpret_cast<float2*>(raw_coords)[tx];
        int2 point{ (int)coord.x, (int)coord.y };
        if (point.x >= 0 && point.x < extent.x && point.y >= 0 && point.y < extent.y)
        {
        seeds[extent.x * point.y + point.x] = make_float4(coord.x, coord.y, 0.f, 0.f);
        }
    }
}

void initJumpFlood(float4 *d_seeds, float *d_coords, int coord_count,
    int2 extent, hipStream_t stream)
{
    dim3 threads{128};
    dim3 blocks1{ (extent.x * extent.y + threads.x - 1) / threads.x};
    dim3 blocks2{ (coord_count + threads.x - 1) / threads.x};

    kernelSetNonSeeds<<< blocks1, threads, 0, stream >>>(d_seeds, extent.x * extent.y);
    checkCuda(hipStreamSynchronize(stream));
    kernelSetSeeds<<< blocks2, threads, 0, stream >>>(
        d_seeds, d_coords, coord_count, extent
    );
    checkCuda(hipStreamSynchronize(stream));
}

__global__ void initSystem(float *coords, size_t particle_count,
    hiprandState *global_states, int2 extent, unsigned seed)
{
    auto particles = reinterpret_cast<float2*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < particle_count)
    {
        auto local_state = global_states[tidx];
        hiprand_init(seed, tidx, 0, &local_state);
        auto rx = extent.x * hiprand_uniform(&local_state);
        auto ry = extent.y * hiprand_uniform(&local_state);
        float2 p{rx, ry};
        particles[tidx] = p;
        global_states[tidx] = local_state;
    }
}

__device__ float clamp(float x, float low, float high)
{
    return fmaxf(low, fminf(high, x));
}

__global__ void integrate2d(float *coords, size_t particle_count,
    hiprandState *global_states, int2 extent)
{
    auto particles = reinterpret_cast<float2*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < particle_count)
    {
        auto local_state = global_states[tidx];
        auto r = hiprand_normal2(&local_state);
        auto p = particles[tidx];
        p.x = clamp(p.x + r.x, 1e-6f, extent.x);
        p.y = clamp(p.y + r.y, 1e-6f, extent.y);
        particles[tidx] = p;
        global_states[tidx] = local_state;
    }
}

JumpFloodProgram::JumpFloodProgram(unsigned point_count, int width, int height):
    element_count{point_count}, extent{width, height}
{
    checkCuda(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
}

void JumpFloodProgram::setInitialState()
{
    checkCuda(hipSetDevice(0));

    //checkCuda(hipMalloc(&_d_distances, dist_size));
    //checkCuda(hipMalloc(&d_coords, sizeof(float2) * element_count));
    checkCuda(hipMallocAsync(&d_states, sizeof(hiprandState) * element_count, stream));

    dim3 threads{128};
    dim3 blocks { (element_count + threads.x - 1) / threads.x};
    initSystem<<<blocks, threads>>>(d_coords, element_count, d_states, extent, 1234);
    checkCuda(hipDeviceSynchronize());

        // Allocate device numeric canvas
        size_t seed_sizes = sizeof(float4) * extent.x * extent.y;
        checkCuda(hipMalloc(&d_grid[0], seed_sizes));
    checkCuda(hipMalloc(&d_grid[1], seed_sizes));
    checkCuda(hipDeviceSynchronize());
        initJumpFlood(d_grid[1], d_coords, element_count, extent, stream);
}

void JumpFloodProgram::cleanup()
{
    checkCuda(hipDeviceSynchronize());
    checkCuda(hipStreamDestroy(stream));
    checkCuda(hipFree(d_grid[0]));
    checkCuda(hipFree(d_grid[1]));
    checkCuda(hipFree(d_states));
    checkCuda(hipFree(d_distances));
    checkCuda(hipFree(d_coords));
    checkCuda(hipDeviceReset());
}

void JumpFloodProgram::runTimestep()
{
    dim3 threads{128};
    dim3 blocks { (element_count + threads.x - 1) / threads.x};

    integrate2d<<< blocks, threads, 0, stream >>>(
        d_coords, element_count, d_states, extent
    );
    checkCuda(hipDeviceSynchronize());

    initJumpFlood(d_grid[1], d_coords, element_count, extent, stream);

    jumpFlood(d_distances, d_grid, extent, stream);
}

int main(int argc, char *argv[])
{
    unsigned point_count = 100;
    size_t iter_count = 10000;
    if (argc >= 2)
    {
        point_count = std::stoul(argv[1]);
    }
    if (argc >= 3)
    {
        iter_count = std::stoul(argv[2]);
    }

    JumpFloodProgram program(point_count, 512, 512);
    try
    {
        MimirEngine engine;
        engine.init(1920, 1080);

        MemoryParams m1;
        m1.layout          = DataLayout::Layout1D;
        m1.element_count.x = program.element_count;
        m1.component_type  = ComponentType::Float;
        m1.channel_count   = 2;
        m1.resource_type   = ResourceType::Buffer;
        auto points = engine.createBuffer((void**)&program.d_coords, m1);

        ViewParams p1;
        p1.element_count = program.element_count;
        p1.extent        = {(unsigned)program.extent.x, (unsigned)program.extent.y, 1};
        p1.data_domain   = DataDomain::Domain2D;
        p1.domain_type   = DomainType::Unstructured;
        p1.view_type     = ViewType::Markers;
        p1.attributes[AttributeType::Position] = *points;
        p1.options.default_color = {0,0,1,1};
        auto v1 = engine.createView(p1);

        MemoryParams m2;
        m2.layout         = DataLayout::Layout2D;
        m2.element_count  = {(uint)program.extent.x, (uint)program.extent.y, 1};
        m2.component_type = ComponentType::Float;
        m2.channel_count  = 1;
        m2.resource_type  = ResourceType::LinearTexture;
        auto image = engine.createBuffer((void**)&program.d_distances, m2);

        ViewParams p2;
        p2.element_count = program.extent.x * program.extent.y;
        p2.data_domain   = DataDomain::Domain2D;
        p2.domain_type   = DomainType::Structured;
        p2.view_type     = ViewType::Image;
        p2.attributes[AttributeType::Color] = *image;
        auto v2 = engine.createView(p2);

        //hipMalloc((void**)&program.d_coords, sizeof(float2) * point_count);
        //hipMalloc((void**)&program.d_distances, sizeof(float) * program.extent.x * program.extent.y);

        program.setInitialState();

        // Start rendering loop
        auto timestep_function = std::bind(&JumpFloodProgram::runTimestep, program);
        engine.display(timestep_function, iter_count);
    }
    catch (const std::exception& e)
    {
        std::cerr << e.what() << std::endl;
        return EXIT_FAILURE;
        program.cleanup();
    }
    program.cleanup();

    return EXIT_SUCCESS;
}
