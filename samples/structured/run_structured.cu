#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <limits> // std::numeric_limits

#include <mimir/mimir.hpp>
#include "validation.hpp" // checkCuda
using namespace mimir;

constexpr float max_distance = std::numeric_limits<float>::max();

__device__
float4 jumpFloodStep(float2 coord, float4 *seeds, int step_length, int2 extent)
{
    float best_dist = max_distance;
    float2 best_coord = make_float2(-1.f, -1.f);

    for (int y = -1; y <= 1; ++y)
    {
        for (int x = -1; x <= 1; ++x)
        {
            int sample_x = coord.x + x * step_length;
            int sample_y = coord.y + y * step_length;
            if (sample_x >= 0 && sample_x < extent.x && sample_y >= 0 && sample_y < extent.y)
            {
                float4 seed = seeds[extent.x * sample_y + sample_x];
                float dist = hypotf(seed.x - coord.x, seed.y - coord.y);

                if ((seed.x != -1.f && seed.y != -1.f) && dist < best_dist)
                {
                    best_dist = dist;
                    best_coord = make_float2(seed.x, seed.y);
                }
            }
        }
    }
    return make_float4(best_coord.x, best_coord.y, 0.f, best_dist);
}

__global__
void kernelJfa(float4 *result, float4 *seeds, const int2 extent, int step_length)
{
    const int tx = blockDim.x * blockIdx.x + threadIdx.x;
    const int ty = blockDim.y * blockIdx.y + threadIdx.y;
    if (tx < extent.x && ty < extent.y)
    {
        float2 coord = make_float2(tx, ty);
        float4 output = jumpFloodStep(coord, seeds, step_length, extent);
        result[extent.x * ty + tx] = output;
    }
}

__global__
void kernelDistanceTransform(float *distances, float4 *seeds, int2 extent)
{
    const int tx = blockDim.x * blockIdx.x + threadIdx.x;
    const int ty = blockDim.y * blockIdx.y + threadIdx.y;

    if (tx < extent.x && ty < extent.y)
    {
        auto grid_idx = extent.x * ty + tx;
        distances[grid_idx] = seeds[grid_idx].w / 200.f;//hypotf(extent.x, extent.y);
    }
}

void jumpFlood(float *distances, float4 *seeds[], int2 extent)
{
    dim3 threads(32, 32);
    dim3 blocks( (extent.x + threads.x - 1) / threads.x,
                 (extent.y + threads.y - 1) / threads.y );

    int out_idx = 0, in_idx = 1;
    for (int k = extent.x / 2; k > 0; k = k >> 1)
    {
        kernelJfa<<< blocks, threads >>>(seeds[out_idx], seeds[in_idx], extent, k);
        checkCuda(hipDeviceSynchronize());
        std::swap(out_idx, in_idx);
    }
    kernelDistanceTransform<<< blocks, threads >>>(distances, seeds[in_idx], extent);
}

__global__
void kernelSetNonSeeds(float4 *seeds, int seed_count)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tx < seed_count)
    {
        seeds[tx] = make_float4(-1.f, -1.f, 0.f, 0.f);
    }
}

__global__
void kernelSetSeeds(float4 *seeds, float *raw_coords, int coord_count, int2 extent)
{
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if (tx < coord_count)
    {
        auto coord = reinterpret_cast<float2*>(raw_coords)[tx];
        int2 point{ (int)coord.x, (int)coord.y };
        if (point.x >= 0 && point.x < extent.x && point.y >= 0 && point.y < extent.y)
        {
            seeds[extent.x * point.y + point.x] = make_float4(coord.x, coord.y, 0.f, 0.f);
        }
    }
}

void initJumpFlood(float4 *d_seeds, float *d_coords, int coord_count,
    int2 extent)
{
    dim3 threads{128};
    dim3 blocks1{ (extent.x * extent.y + threads.x - 1) / threads.x};
    dim3 blocks2{ (coord_count + threads.x - 1) / threads.x};

    kernelSetNonSeeds<<< blocks1, threads >>>(d_seeds, extent.x * extent.y);
    checkCuda(hipDeviceSynchronize());
    kernelSetSeeds<<< blocks2, threads >>>(d_seeds, d_coords, coord_count, extent);
    checkCuda(hipDeviceSynchronize());
}

__global__ void initSystem(float *coords, size_t particle_count,
    hiprandState *global_states, int2 extent, unsigned seed)
{
    auto particles = reinterpret_cast<float2*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < particle_count)
    {
        auto local_state = global_states[tidx];
        hiprand_init(seed, tidx, 0, &local_state);
        auto rx = extent.x * hiprand_uniform(&local_state);
        auto ry = extent.y * hiprand_uniform(&local_state);
        float2 p{rx, ry};
        particles[tidx] = p;
        global_states[tidx] = local_state;
    }
}

__device__ float clamp(float x, float low, float high)
{
    return fmaxf(low, fminf(high, x));
}

__global__ void integrate2d(float *coords, size_t particle_count,
    hiprandState *global_states, int2 extent)
{
    auto particles = reinterpret_cast<float2*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < particle_count)
    {
        auto local_state = global_states[tidx];
        auto r = hiprand_normal2(&local_state);
        auto p = particles[tidx];
        p.x = clamp(p.x + r.x, 1e-6f, extent.x);
        p.y = clamp(p.y + r.y, 1e-6f, extent.y);
        particles[tidx] = p;
        global_states[tidx] = local_state;
    }
}

int main(int argc, char *argv[])
{
    float *d_distances    = nullptr;
    float *d_coords       = nullptr;
    float4 *d_grid[2]     = {nullptr, nullptr};
    hiprandState *d_states = nullptr;
    int2 extent           = {512, 512};

    unsigned seed_count = 100;
    size_t iter_count = 10000;
    if (argc >= 2) seed_count = std::stoul(argv[1]);
    if (argc >= 3) iter_count  = std::stoul(argv[2]);

    Engine engine = nullptr;
    createEngine(1920, 1080, &engine);

    AllocHandle seeds = nullptr, field = nullptr;
    allocLinear(engine, (void**)&d_coords, sizeof(float2) * seed_count, &seeds);
    allocLinear(engine, (void**)&d_distances, sizeof(float) * extent.x * extent.y, &field);

    ViewHandle v1 = nullptr, v2 = nullptr;
    ViewDescription desc;
    desc.element_count = seed_count;
    desc.view_type     = ViewType::Markers;
    desc.domain_type   = DomainType::Domain2D;
    desc.extent        = {(unsigned)extent.x, (unsigned)extent.y, 1};
    desc.attributes[AttributeType::Position] = {
        .source = seeds,
        .size   = seed_count,
        .format = FormatDescription::make<float2>(),
    };
    createView(engine, &desc, &v1);
    //v1->default_color = {0,0,1,1};

    desc.element_count = extent.x * extent.y;
    desc.view_type     = ViewType::Voxels;
    desc.attributes[AttributeType::Position] =
        makeStructuredGrid(engine, desc.extent, {0.f,0.f,0.4999f});
    desc.attributes[AttributeType::Color] = {
        .source = field,
        .size   = desc.element_count,
        .format = FormatDescription::make<float>(),
    };
    createView(engine, &desc, &v2);
    v2->default_size = 1.f;

    //checkCuda(hipMalloc(&_d_distances, dist_size));
    //checkCuda(hipMalloc(&d_coords, sizeof(float2) * element_count));
    checkCuda(hipMalloc(&d_states, sizeof(hiprandState) * seed_count));

    dim3 threads{128};
    dim3 blocks { (seed_count + threads.x - 1) / threads.x};
    initSystem<<<blocks, threads>>>(d_coords, seed_count, d_states, extent, 1234);
    checkCuda(hipDeviceSynchronize());

    // Allocate device numeric canvas
    size_t seed_sizes = sizeof(float4) * extent.x * extent.y;
    checkCuda(hipMalloc(&d_grid[0], seed_sizes));
    checkCuda(hipMalloc(&d_grid[1], seed_sizes));
    checkCuda(hipDeviceSynchronize());
    initJumpFlood(d_grid[1], d_coords, seed_count, extent);

    // Start rendering loop
    auto timestep_function = [&]{
        dim3 threads{128};
        dim3 blocks { (seed_count + threads.x - 1) / threads.x};

        integrate2d<<< blocks, threads >>>(d_coords, seed_count, d_states, extent);
        checkCuda(hipDeviceSynchronize());
        initJumpFlood(d_grid[1], d_coords, seed_count, extent);
        jumpFlood(d_distances, d_grid, extent);
        checkCuda(hipDeviceSynchronize());
    };
    display(engine, timestep_function, iter_count);

    checkCuda(hipFree(d_grid[0]));
    checkCuda(hipFree(d_grid[1]));
    checkCuda(hipFree(d_states));
    checkCuda(hipFree(d_distances));
    checkCuda(hipFree(d_coords));
    destroyEngine(engine);

    return EXIT_SUCCESS;
}
