#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h" // stbi_load

#include <mimir/mimir.hpp>
#include <mimir/validation.hpp> // checkCuda
using namespace mimir;
using namespace mimir::validation; // checkCuda

int main(int argc, char *argv[])
{
    char *filepath = nullptr;
    if (argc == 2)
    {
        filepath = argv[1];
    }
    else
    {
        printf("Usage: ./image path/to/image\n");
        return EXIT_FAILURE;
    }

    uchar4 *d_pixels = nullptr;
    int width, height, chans;
    auto h_pixels = stbi_load(filepath, &width, &height, &chans, STBI_rgb_alpha);
    if (!h_pixels)
    {
        printf("failed to load texture image: %s\n", filepath);
        return EXIT_FAILURE;
    }

    MimirEngine engine;
    engine.init(1920, 1080);

    MemoryParams m;
    m.layout         = DataLayout::Layout2D;
    m.element_count  = {(uint)width, (uint)height, 1};
    m.component_type = ComponentType::Char;
    m.channel_count  = 4;
    m.resource_type  = ResourceType::LinearTexture;
    auto pixels = engine.createBuffer((void**)&d_pixels, m);

    ViewParams params;
    params.element_count = width * height;
    params.data_domain   = DataDomain::Domain2D;
    params.domain_type   = DomainType::Structured;
    params.view_type     = ViewType::Image;
    params.attributes[AttributeType::Color] = *pixels;
    /*params.options.external_shaders = {
        {"shaders/texture_vertex2dMain.spv", VK_SHADER_STAGE_VERTEX_BIT},
        {"shaders/texture_frag2d_Float4.spv", VK_SHADER_STAGE_FRAGMENT_BIT}
    };*/
    engine.createView(params);

    auto tex_size = sizeof(uchar4) * width * height;
    checkCuda(hipMemcpy(d_pixels, h_pixels, tex_size, hipMemcpyHostToDevice));
    stbi_image_free(h_pixels);

    engine.displayAsync();
    checkCuda(hipFree(d_pixels));

    return EXIT_SUCCESS;
}
