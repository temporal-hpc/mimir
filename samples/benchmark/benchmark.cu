#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h> // cooperative_groups::{sync, this_thread_block}
#include <iostream>
#include <random>
#include <string> // std::stoul

#include <mimir/mimir.hpp>
#include "validation.hpp" // checkCuda

#include "nvmlPower.hpp"

using namespace mimir;
namespace cg = cooperative_groups;

struct NBodyParams
{
    float time_step;
    float cluster_scale;
    float velocity_scale;
    float softening;
    float damping;
    float point_size;
    float x, y, z;

    void print()
    {
        printf("{ %f, %f, %f, %f, %f, %f, %f, %f, %f },\n", time_step,
            cluster_scale, velocity_scale, softening, damping, point_size, x, y, z
        );
    }
};

NBodyParams demo_params[] = {
    {0.016f, 1.54f, 8.0f, 0.1f, 1.0f, 1.0f, 0, -2, -100},
    {0.016f, 0.68f, 20.0f, 0.1f, 1.0f, 0.8f, 0, -2, -30},
    {0.0006f, 0.16f, 1000.0f, 1.0f, 1.0f, 0.07f, 0, 0, -1.5f},
    {0.0006f, 0.16f, 1000.0f, 1.0f, 1.0f, 0.07f, 0, 0, -1.5f},
    {0.0019f, 0.32f, 276.0f, 1.0f, 1.0f, 0.07f, 0, 0, -5},
    {0.0016f, 0.32f, 272.0f, 0.145f, 1.0f, 0.08f, 0, 0, -5},
    {0.016f, 6.040f, 0.f, 1.f, 1.f, 0.760f, 0, 0, -50},
};

__constant__ float softening_squared;

hipError_t setSofteningSquared(float value)
{
    return hipMemcpyToSymbol(HIP_SYMBOL(softening_squared), &value, sizeof(float), 0, hipMemcpyHostToDevice);
}

struct SharedMemory {
    __device__ inline operator float4 *()
    {
        extern __shared__ int __smem[];
        return (float4 *)__smem;
    }

    __device__ inline operator const float4 *() const
    {
        extern __shared__ int __smem[];
        return (float4 *)__smem;
    }
};

struct DeviceData
{
    float4 *dPos[2];  // mapped host pointers
    float4 *dVel;
};

__device__ float3 bodyBodyInteraction(float3 ai, float4 bi, float4 bj)
{
    float3 r;

    // r_ij  [3 FLOPS]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;

    // dist_sqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
    float dist_sqr = r.x * r.x + r.y * r.y + r.z * r.z;
    dist_sqr += softening_squared;

    // inv_dist_cube =1/dist_sqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    float inv_dist = rsqrtf(dist_sqr);
    float inv_dist_cube = inv_dist * inv_dist * inv_dist;

    // s = m_j * inv_dist_cube [1 FLOP]
    float s = bj.w * inv_dist_cube;

    // a_i =  a_i + s * r_ij [6 FLOPS]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;

    return ai;
}

__device__ float3 computeBodyAccel(float4 body_pos, float4 *positions, int num_tiles,
    cg::thread_block cta)
{
    float4 *shared_pos = SharedMemory();
    float3 acc = {0.0f, 0.0f, 0.0f};

    for (int tile = 0; tile < num_tiles; tile++)
    {
        shared_pos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];
        cg::sync(cta);

// This is the "tile_calculation" from the GPUG3 article.
#pragma unroll 128
        for (unsigned int counter = 0; counter < blockDim.x; counter++)
        {
            acc = bodyBodyInteraction(acc, body_pos, shared_pos[counter]);
        }
        cg::sync(cta);
    }

    return acc;
}

__global__ void integrateBodies(float4 *__restrict__ new_pos, float4 *__restrict__ old_pos,
    float4 *vel, unsigned int body_count, float delta_time, float damping, int num_tiles)
{
    // Handle to thread block group
    cg::thread_block cta = cg::this_thread_block();
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= body_count) { return; }

    float4 position = old_pos[index];
    float3 accel = computeBodyAccel(position, old_pos, num_tiles, cta);

    // acceleration = force / mass;
    // new velocity = old velocity + acceleration * delta_time
    // note we factor out the body's mass from the equation, here and in bodyBodyInteraction
    // (because they cancel out). Thus here force == acceleration
    float4 velocity = vel[index];

    velocity.x += accel.x * delta_time;
    velocity.y += accel.y * delta_time;
    velocity.z += accel.z * delta_time;

    velocity.x *= damping;
    velocity.y *= damping;
    velocity.z *= damping;

    // new position = old position + velocity * delta_time
    position.x += velocity.x * delta_time;
    position.y += velocity.y * delta_time;
    position.z += velocity.z * delta_time;

    // store new position and velocity
    new_pos[index] = position;
    vel[index] = velocity;
}

void integrateNbodySystem(DeviceData device_data, unsigned int current_read, float delta_time,
    float damping, unsigned int body_count, int block_size)
{
    int num_blocks = (body_count + block_size - 1) / block_size;
    int num_tiles = (body_count + block_size - 1) / block_size;
    int shmem_size = block_size * 4 * sizeof(float);  // 4 floats for pos

    integrateBodies<<<num_blocks, block_size, shmem_size>>>(
        device_data.dPos[1 - current_read],
        device_data.dPos[current_read],
        device_data.dVel,
        body_count, delta_time, damping, num_tiles
    );

    // check if kernel invocation generated an error
    checkCuda(hipGetLastError());
}

enum class NBodyConfig { Random, Shell, Expand };

inline float normalize(float3 &vector)
{
    float dist = sqrtf(vector.x * vector.x + vector.y * vector.y + vector.z * vector.z);
    if (dist > 1e-6)
    {
        vector.x /= dist;
        vector.y /= dist;
        vector.z /= dist;
    }

    return dist;
}

inline float dot(float3 v0, float3 v1)
{
    return v0.x * v1.x + v0.y * v1.y + v0.z * v1.z;
}

inline float3 cross(float3 v0, float3 v1)
{
    float3 rt;
    rt.x = v0.y * v1.z - v0.z * v1.y;
    rt.y = v0.z * v1.x - v0.x * v1.z;
    rt.z = v0.x * v1.y - v0.y * v1.x;
    return rt;
}

void randomizeBodies(NBodyConfig config, float *pos, float *vel, float *color,
    float cluster_scale, float velocity_scale, int body_count, bool vec4vel)
{
    std::mt19937 rng(12345);
    std::uniform_real_distribution<float> rand_pos(-1, 1);
    switch (config)
    {
        default:
        case NBodyConfig::Random:
        {
            float scale = cluster_scale * std::max<float>(1.0f, body_count / (1024.0f));
            float vscale = velocity_scale * scale;

            int p = 0, v = 0;
            int i = 0;

            while (i < body_count)
            {
                float3 point;
                // const int scale = 16;
                point.x = rand_pos(rng);
                point.y = rand_pos(rng);
                point.z = rand_pos(rng);
                float lenSqr = dot(point, point);

                if (lenSqr > 1) continue;

                float3 velocity;
                velocity.x = rand_pos(rng);
                velocity.y = rand_pos(rng);
                velocity.z = rand_pos(rng);
                lenSqr = dot(velocity, velocity);

                if (lenSqr > 1) continue;

                pos[p++] = point.x * scale;  // pos.x
                pos[p++] = point.y * scale;  // pos.y
                pos[p++] = point.z * scale;  // pos.z
                pos[p++] = 1.0f;             // mass

                vel[v++] = velocity.x * vscale;  // pos.x
                vel[v++] = velocity.y * vscale;  // pos.x
                vel[v++] = velocity.z * vscale;  // pos.x

                if (vec4vel) vel[v++] = 1.0f;  // inverse mass

                i++;
            }
        } break;

        case NBodyConfig::Shell:
        {
            float scale = cluster_scale;
            float vscale = scale * velocity_scale;
            float inner = 2.5f * scale;
            float outer = 4.0f * scale;

            int p = 0, v = 0;
            int i = 0;

            while (i < body_count)
            {
                float x, y, z;
                x = rand_pos(rng);
                y = rand_pos(rng);
                z = rand_pos(rng);

                float3 point = {x, y, z};
                float len = normalize(point);
                if (len > 1) { continue; }

                pos[p++] = point.x * (inner + (outer - inner) * rand() / (float)RAND_MAX);
                pos[p++] = point.y * (inner + (outer - inner) * rand() / (float)RAND_MAX);
                pos[p++] = point.z * (inner + (outer - inner) * rand() / (float)RAND_MAX);
                pos[p++] = 1.0f;

                x = 0.0f;  // * (rand() / (float) RAND_MAX * 2 - 1);
                y = 0.0f;  // * (rand() / (float) RAND_MAX * 2 - 1);
                z = 1.0f;  // * (rand() / (float) RAND_MAX * 2 - 1);
                float3 axis = {x, y, z};
                normalize(axis);

                if (1 - dot(point, axis) < 1e-6)
                {
                    axis.x = point.y;
                    axis.y = point.x;
                    normalize(axis);
                }

                // if (point.y < 0) axis = scalevec(axis, -1);
                float3 vv = {(float)pos[4 * i], (float)pos[4 * i + 1], (float)pos[4 * i + 2]};
                vv = cross(vv, axis);
                vel[v++] = vv.x * vscale;
                vel[v++] = vv.y * vscale;
                vel[v++] = vv.z * vscale;

                if (vec4vel) { vel[v++] = 1.0f; }

                i++;
            }
        } break;

        case NBodyConfig::Expand:
        {
            float scale = cluster_scale * body_count / (1024.f);

            if (scale < 1.0f) { scale = cluster_scale; }
            float vscale = scale * velocity_scale;
            int p = 0, v = 0;

            for (int i = 0; i < body_count;)
            {
                float3 point;
                point.x = rand_pos(rng);
                point.y = rand_pos(rng);
                point.z = rand_pos(rng);

                float lenSqr = dot(point, point);
                if (lenSqr > 1) { continue; }

                pos[p++] = point.x * scale;   // pos.x
                pos[p++] = point.y * scale;   // pos.y
                pos[p++] = point.z * scale;   // pos.z
                pos[p++] = 1.0f;              // mass
                vel[v++] = point.x * vscale;  // pos.x
                vel[v++] = point.y * vscale;  // pos.x
                vel[v++] = point.z * vscale;  // pos.x

                if (vec4vel) vel[v++] = 1.0f;  // inverse mass

                i++;
            }
        } break;
    }

    if (color != nullptr)
    {
        std::uniform_real_distribution<float> rand_color(0, 1);
        int v = 0;
        for (int i = 0; i < body_count; i++) {
            color[v++] = rand_color(rng);
            color[v++] = rand_color(rng);
            color[v++] = rand_color(rng);
            color[v++] = 1.0f;
        }
    }
}

int main(int argc, char *argv[])
{
    // CUDA initialization
    const int device_id = 0;
    checkCuda(hipSetDevice(device_id));
    const unsigned int block_size = 256;

    // Default experiment parameters
    int width                = 1920;
    int height               = 1080;
    unsigned int body_count  = 77824;
    int iter_count           = 1000000;
    PresentMode present_mode = PresentMode::Immediate;
    int target_fps           = 0;
    bool enable_sync         = true;
    bool use_interop         = true;
    NBodyParams params       = demo_params[3];

    // Parse parameters from command line
    if (argc >= 3) { width = std::stoi(argv[1]); height = std::stoi(argv[2]); }
    if (argc >= 4) body_count   = std::stoul(argv[3]);
    if (argc >= 5) iter_count   = std::stoi(argv[4]);
    if (argc >= 6) present_mode = static_cast<PresentMode>(std::stoi(argv[5]));
    if (argc >= 7) target_fps   = std::stoi(argv[6]);
    if (argc >= 8) enable_sync  = static_cast<bool>(std::stoi(argv[7]));
    if (argc >= 9) use_interop  = static_cast<bool>(std::stoi(argv[8]));

    // Determine execution mode for benchmarking and write CSV column names
    std::string mode;
    if (width == 0 && height == 0) mode = use_interop? "mimir" : "no_disp";
    else mode = enable_sync? "sync" : "desync";

    bool display = true;
    if (width == 0 || height == 0)
    {
        width = height = 10;
        display = false;
    }

    ViewerOptions options;
    options.window.size   = {width,height}; // Starting window size
    options.show_metrics  = false; // Show metrics window in GUI
    options.report_period = 0; // Print relevant usage stats every N seconds
    options.background_color = {0.f, 0.f, 0.f, 1.f};
    options.present = {
        .mode        = present_mode,
        .enable_sync = enable_sync,
        .target_fps  = target_fps,
    };
    EngineHandle engine = nullptr;
    createEngine(options, &engine);
    setCameraPosition(engine, {params.x, params.y, params.z});

    auto nbody_memsize = sizeof(float4) * body_count;
    DeviceData data;
    checkCuda(hipMalloc((void**)&data.dVel, nbody_memsize));

    mimir::ViewHandle views[2];
    if (use_interop)
    {
        mimir::AllocHandle allocs[2];
        allocLinear(engine, (void**)&data.dPos[0], nbody_memsize, &allocs[0]);
        allocLinear(engine, (void**)&data.dPos[1], nbody_memsize, &allocs[1]);

        ViewDescription desc{
            .layout      = Layout::make(body_count),
            .view_type   = ViewType::Markers,
            .domain_type = DomainType::Domain3D,
            .attributes  = {
                { AttributeType::Position, {
                    .source = allocs[0],
                    .size   = body_count,
                    .format = FormatDescription::make<float4>(),
                    .indices = {},
                    .index_size = 0,
                }}
            },
            .visible       = true,
            .default_color = {1.f, 1.f, 1.f, 1.f},
            .default_size  = params.point_size / 2.f,
            .scale         = {1.f, 1.f, 1.f},
        };
        createView(engine, &desc, &views[0]);

        desc.visible = false;
        desc.attributes[AttributeType::Position].source = allocs[1];
        createView(engine, &desc, &views[1]);
    }
    else // Run the simulation without display
    {
        checkCuda(hipMalloc((void**)&data.dPos[0], nbody_memsize));
        checkCuda(hipMalloc((void**)&data.dPos[1], nbody_memsize));
    }

    // Initialize simulation
    unsigned int current_read  = 0;
    unsigned int current_write = 1;

    NBodyConfig config = NBodyConfig::Shell;
    setSofteningSquared(params.softening);
    float *h_pos = new float[body_count * 4];
    float *h_vel = new float[body_count * 4];
    randomizeBodies(config, h_pos, h_vel, nullptr, params.cluster_scale, params.velocity_scale, body_count, true);
    checkCuda(hipMemcpy(data.dPos[current_read], h_pos, nbody_memsize, hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(data.dVel, h_vel, nbody_memsize, hipMemcpyHostToDevice));
    delete[] h_pos;
    delete[] h_vel;

    // Start display and measurements
    setCameraPosition(engine, {1.f, 1.f, -3.f});
    GPUPowerBegin("gpu", 100);
    if (display) displayAsync(engine);
    std::cin.get();

    // Main simulation loop
    for (int i = 0; i < iter_count; ++i)
    {
        if (display) prepareViews(engine);
        integrateNbodySystem(data, current_read, params.time_step, params.damping, body_count, block_size);
        std::swap(current_read, current_write);
        if (display)
        {
            toggleVisibility(views[0]);
            toggleVisibility(views[1]);
            updateViews(engine);
        }
    }

    // Retrieve metrics
    printf("%s,%u,", mode.c_str(), body_count);
    getMetrics(engine);

    // Nvml memory report
    {
        nvmlMemory_v2_t meminfo;
        meminfo.version = (unsigned int)(sizeof(nvmlMemory_v2_t) | (2 << 24U));
        nvmlDeviceGetMemoryInfo_v2(getNvmlDevice(), &meminfo);

        constexpr double gigabyte = 1024.0 * 1024.0 * 1024.0;
        double freemem = meminfo.free / gigabyte;
        double reserved = meminfo.reserved / gigabyte;
        double totalmem = meminfo.total / gigabyte;
        double usedmem = meminfo.used / gigabyte;
        printf("%lf,%lf,", freemem, usedmem);
    }
    GPUPowerEnd();

    // Cleanup
    exit(engine);
    destroyEngine(engine);
    checkCuda(hipFree(data.dPos[0]));
    checkCuda(hipFree(data.dPos[1]));
    checkCuda(hipFree(data.dVel));

    return EXIT_SUCCESS;
}
