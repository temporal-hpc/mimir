#include "hip/hip_runtime.h"
#include <experimental/source_location> // std::source_location
#include <hiprand/hiprand_kernel.h>
#include <chrono> // std::chrono
#include <string> // std::stoul

#include "nvmlPower.hpp"
#include <pcl/point_types.h>
#include <pcl/visualization/cloud_viewer.h>

using chrono_tp = std::chrono::time_point<std::chrono::high_resolution_clock>;
using source_location = std::experimental::source_location;

constexpr void checkCuda(hipError_t code, bool panic = true,
    source_location src = source_location::current())
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA assertion: %s in function %s at %s(%d)\n",
            hipGetErrorString(code), src.function_name(), src.file_name(), src.line()
        );
        if (panic)
        {
            exit(EXIT_FAILURE);
        }
    }
}

__global__ void initSystem(float *coords, size_t point_count,
    hiprandState *global_states, uint3 extent, unsigned seed)
{
    auto points = reinterpret_cast<float4*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < point_count)
    {
        auto local_state = global_states[tidx];
        hiprand_init(seed, tidx, 0, &local_state);
        auto rx = extent.x * hiprand_uniform(&local_state);
        auto ry = extent.y * hiprand_uniform(&local_state);
        auto rz = extent.z * hiprand_uniform(&local_state);
        points[tidx] = {rx, ry, rz, 0};
        global_states[tidx] = local_state;
    }
}

__global__ void integrate3d(float *coords, size_t point_count,
    hiprandState *global_states, uint3 extent)
{
    auto points = reinterpret_cast<float4*>(coords);
    auto tidx = blockDim.x * blockIdx.x + threadIdx.x;
    if (tidx < point_count)
    {
        auto local_state = global_states[tidx];
        auto p = points[tidx];
        p.x += hiprand_normal(&local_state);
        if (p.x > extent.x) p.x = extent.x;
        if (p.x < 0) p.x = 0;
        p.y += hiprand_normal(&local_state);
        if (p.y > extent.x) p.y = extent.y;
        if (p.y < 0) p.y = 0;
        p.z += hiprand_normal(&local_state);
        if (p.z > extent.z) p.z = extent.z;
        if (p.z < 0) p.z = 0;
        points[tidx] = p;
        global_states[tidx] = local_state;
    }
}

int main(int argc, char *argv[])
{
    float *d_coords       = nullptr;
    hiprandState *d_states = nullptr;
    unsigned block_size   = 256;
    unsigned seed         = 123456;
    uint3 extent          = {200, 200, 200};

    // Default values for this program
    unsigned int point_count = 100;
    int iter_count = 10000;
    if (argc >= 2) point_count = std::stoul(argv[1]);
    if (argc >= 3) iter_count = std::stoi(argv[2]);

    checkCuda(hipMalloc((void**)&d_coords, sizeof(float4) * point_count));
    checkCuda(hipMalloc(&d_states, sizeof(hiprandState) * point_count));
    unsigned grid_size = (point_count + block_size - 1) / block_size;
    initSystem<<<grid_size, block_size>>>(d_coords, point_count, d_states, extent, seed);
    checkCuda(hipDeviceSynchronize());

    // Initialize point cloud structure on host memory
    pcl::PointCloud<pcl::PointXYZ>::Ptr cloud(new pcl::PointCloud<pcl::PointXYZ>());
    //pcl::PointCloud<pcl::PointXYZ> cloud;
    cloud->width = point_count;
    cloud->height = 1;
    cloud->is_dense = true;
    cloud->resize(point_count);
    checkCuda(hipMemcpy(cloud->points.data(), d_coords,
        sizeof(float4) * point_count, hipMemcpyDeviceToHost)
    );

    pcl::visualization::PCLVisualizer::Ptr viewer(new pcl::visualization::PCLVisualizer ("3D Viewer", true));
    viewer->setBackgroundColor (0.0, 0.0, 0.0);
    viewer->addPointCloud<pcl::PointXYZ> (cloud, "points3d");
    viewer->setPointCloudRenderingProperties (pcl::visualization::PCL_VISUALIZER_POINT_SIZE, 1, "points3d");
    viewer->addCoordinateSystem (1.0);
    viewer->initCameraParameters ();
    viewer->setSize(1920, 1080);
    viewer->setCameraPosition(-199.419, 277.404, 279.833, 56.991, 13.6156, 47.422, 0, 0, 1);
    //viewer->setFullScreen(true); // Produces segmentation fault

    // Main loop variables
    int iter_idx = 0;
    float total_graphics_time = 0;
    chrono_tp last_time = {};
    std::array<float,240> frame_times{};
    size_t total_frame_count = 0;

    GPUPowerBegin("gpu", 100);
    while (!viewer->wasStopped())
    {
        if (iter_idx < iter_count)
        {
            // Measure frame time
            static chrono_tp start_time = std::chrono::high_resolution_clock::now();
            chrono_tp current_time = std::chrono::high_resolution_clock::now();
            if (iter_idx == 0)
            {
                last_time = start_time;
            }
            float frame_time = std::chrono::duration<float, std::chrono::seconds::period>(current_time - last_time).count();

            viewer->removeAllPointClouds();

            integrate3d<<<grid_size, block_size>>>(d_coords, point_count, d_states, extent);
            checkCuda(hipDeviceSynchronize());

            checkCuda(hipMemcpy(cloud->points.data(), d_coords,
                sizeof(float4) * point_count, hipMemcpyDeviceToHost)
            );

            viewer->addPointCloud(cloud, "points3d");
            viewer->setPointCloudRenderingProperties (pcl::visualization::PCL_VISUALIZER_POINT_SIZE, 1, "points3d");
            viewer->spinOnce(100);

            iter_idx++;
            total_frame_count++;
            total_graphics_time += frame_time;
            frame_times[iter_idx % frame_times.size()] = frame_time;
            last_time = current_time;
        } else {
            auto frame_sample_size = std::min(frame_times.size(), total_frame_count);
            float total_frame_time = 0;
            for (size_t i = 0; i < frame_sample_size; ++i) total_frame_time += frame_times[i];
            auto framerate = frame_times.size() / total_frame_time;

            // Nvml memory report
            nvmlMemory_v2_t meminfo;
            meminfo.version = (unsigned int)(sizeof(nvmlMemory_v2_t) | (2 << 24U));
            nvmlDeviceGetMemoryInfo_v2(getNvmlDevice(), &meminfo);

            constexpr double gigabyte = 1024.0 * 1024.0 * 1024.0;
            double nvml_free = meminfo.free / gigabyte;
            double nvml_reserved = meminfo.reserved / gigabyte;
            double nvml_total = meminfo.total / gigabyte;
            double nvml_used = meminfo.used / gigabyte;

            auto gpu = GPUPowerEnd();

            checkCuda(hipFree(d_states));
            checkCuda(hipFree(d_coords));

            printf("PCL,FHD,%d,%f,%f,%f,%f,%f,%f,%f,%f\n",
                point_count,
                framerate,
                gpu.average_power,
                gpu.total_energy,
                gpu.total_time,
                nvml_free,
                nvml_reserved,
                nvml_total,
                nvml_used
            );

            // Flush output before segmentation fault
            std::flush(std::cout);

            viewer->close(); // This causes segfault :(
        }
        /*std::vector<pcl::visualization::Camera> cam;
        viewer->getCameras(cam);
        printf("pos:   %f %f %f\nview:  %f %f %f\nfocal: %f %f %f\n",
            cam[0].pos[0], cam[0].pos[1], cam[0].pos[2],
            cam[0].view[0], cam[0].view[1], cam[0].view[2],
            cam[0].focal[0], cam[0].focal[1], cam[0].focal[2]
        );*/
    }

    return EXIT_SUCCESS;
}
